#include "hip/hip_runtime.h"
#include <cuda/std/barrier>

using barrier = cuda::std::barrier<>;
__managed__ cuda::std::aligned_storage<sizeof(barrier), alignof(barrier)>::type b_;

__global__ void test()
{
    auto& b = reinterpret_cast<barrier&>(b_);
    for(int i = 0;i < 1024; ++i)
        b.arrive_and_wait();
}

int main()
{
    new (&b_) cuda::std::barrier<>(256);

    test<<<32, 8>>>();
    hipDeviceSynchronize();

    return 0;
}
